#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>

#define N 10

__global__ void add(float *a, float *b, float *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void cuda_check(hipError_t status) {
  if (status != hipSuccess) {
    std::cout << "Error could not allocate memory result " << status << std::endl;
    exit(1);
  }
}

int main(void) {
  std::vector<float> vec_a; 
  std::vector<float> vec_b; 
  std::vector<float> vec_c;

  float *a, *b, *c;
  float *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);
  
  cuda_check(hipHostAlloc((void **)&a, size, hipHostMallocPortable));
  cuda_check(hipHostAlloc((void **)&b, size, hipHostMallocPortable));
  cuda_check(hipHostAlloc((void **)&c, size, hipHostMallocPortable));
  
//  a = new int[N];
//  b = new int[N];
//  c = new int[N];

  for(int i = 0; i < N; ++i) {
    vec_a.push_back(i);
    vec_b.push_back(i);
    //a[i] = i;
    //b[i] = i;
  }
  
  memcpy(a, &vec_a[0], size);
  memcpy(b, &vec_b[0], size);


  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<N, 1>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i = 0; i <  N; ++i) {
    std::cout << c[i] << std::endl;
  }

  //free(a); free(b); free(c);
  hipHostFree(a); hipHostFree(b); hipHostFree(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
